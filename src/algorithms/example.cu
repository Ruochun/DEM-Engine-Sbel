#include <hipcub/hipcub.hpp>
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>
#include <cub/util_debug.cuh>
#include <iostream>
#include <core/utils/GpuError.h>
#include "example.h"

void cubIntRunLength(int in[], int in_size, int unique_out[], int count_out[], int run_out[]){

    int* tmp = NULL;
    GPU_CALL(hipMallocManaged(&tmp, 1));

    hipcub::CachingDeviceAllocator g_allocator(true); 

    // Set up device arrays
    int *d_in;
    g_allocator.DeviceAllocate((void **)&d_in, sizeof(int) * in_size);
    hipMemcpy(d_in, in, sizeof(int) * in_size, hipMemcpyHostToDevice);

    // Set up device arrays
    int *d_unique;
    g_allocator.DeviceAllocate((void **)&d_unique, sizeof(int) * in_size);

    // Set up device keys
    int *d_count;
    g_allocator.DeviceAllocate((void **)&d_count, sizeof(int) * in_size);

    // set up res key arrays
    int *d_run_out;
    g_allocator.DeviceAllocate((void **)&d_run_out, sizeof(int) * in_size);

    void *d_temp_storage_1 = NULL;
    size_t temp_storage_bytes_1 = 0;

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage_1, temp_storage_bytes_1,
                                     d_in, d_unique, d_count, d_run_out, in_size);

    g_allocator.DeviceAllocate(&d_temp_storage_1, temp_storage_bytes_1);

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage_1, temp_storage_bytes_1,
                                     d_in, d_unique, d_count, d_run_out, in_size);

    // std::cout << "d_unique: " << std::endl;
    // for (int i = 0; i < 7; i++) {
    //     std::cout << d_unique[i] << " ";
    // }
    // std::cout << std::endl;

    // std::cout << "d_count: " << std::endl;
    // for (int i = 0; i < 7; i++) {
    //     std::cout << d_count[i] << " ";
    // }
    // std::cout << std::endl;

    // std::cout << "d_run_out: " << std::endl;
    // for (int i = 0; i < 7; i++) {
    //     std::cout << d_run_out[i] << " ";
    // }
    // std::cout << std::endl;


    hipDeviceSynchronize();


    hipMemcpy(unique_out, d_unique, sizeof(int) * in_size, hipMemcpyDeviceToHost);
    hipMemcpy(count_out, d_count, sizeof(int) * in_size, hipMemcpyDeviceToHost);
    hipMemcpy(run_out, d_run_out, sizeof(int) * in_size, hipMemcpyDeviceToHost);

    // Cleanup
    if (d_in)
        CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_unique)
        CubDebugExit(g_allocator.DeviceFree(d_unique));
    if (d_count)
        CubDebugExit(g_allocator.DeviceFree(d_count));
    if (d_run_out)
        CubDebugExit(g_allocator.DeviceFree(d_run_out));
    if (d_temp_storage_1)
        CubDebugExit(g_allocator.DeviceFree(d_temp_storage_1));



}